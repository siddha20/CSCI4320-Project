
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "hiprand/hiprand_kernel.h"

#define WORD_LENGTH 5

int node_count;
int size;
int rank;

__device__ int mutex = 0;

void cuda_init(int l_rank, int l_size, int l_node_count) {

    node_count = l_node_count;
    size = l_size;
    rank = l_rank;

    int device_count;
    hipError_t cE;
    
    // Get cuda device count
    if((cE = hipGetDeviceCount(&device_count)) != hipSuccess) {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, device_count);
        exit(-1);
    }

    // Assign rank to cuda device.
    if((cE = hipSetDevice(rank % device_count)) != hipSuccess) {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", rank, (rank % device_count), cE);
        exit(-1);
    }
    
    printf("Rank %d attached to CUDA device %d.\n", rank, (rank % device_count));

}

__global__ void test_write_kernel(int* data) {
    data[threadIdx.x] = threadIdx.x;
}

__global__ void setup_kernel(hiprandState* state, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, state + idx);
}

__global__ void generate_graph_kernel(int* data, int* write_count, int buf_len, hiprandState* state, float p, 
                                      int node_count, int nodes_per_rank, int start_node, int end_node) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = start_node + idx;

    // printf("write count: %d, buf_len %d\n", *write_count, buf_len);
    if (*write_count >= buf_len) return;

    if (i < end_node) {
        // Create edges i --> j
        for (int j = 0; j < node_count; j++) {
            float rand = hiprand_uniform(state + idx);
            if (rand <= p) {
                int old = 1;
                while (old) {
                    old = atomicCAS(&mutex, 0, 1); // lock
                    if (old == 0){
                        data[(*write_count)] = i;
                        data[(*write_count) + 1] = j;
                        // printf("edge %d --> %d, %f\n", data[(*write_count)], data[(*write_count) + 1], rand);
                        (*write_count) += 2;
                        // printf("write count %d\n", *write_count);
                        __threadfence();

                        atomicExch(&mutex, 0); // unlock
                    }
                }
            }
        }
        __syncthreads();
    }
}


void generate_graph(int* h_buf, int buf_len, int* h_write_count, float h_p) {

    int nodes_per_rank = (node_count/size) + ((node_count % size) != 0);
    int start_node = rank * nodes_per_rank;
    int end_node = (start_node + nodes_per_rank) < node_count ? start_node + nodes_per_rank : node_count;

    // Allocate memory on device. 
    int* d_buf;
    hipMalloc(&d_buf, buf_len * sizeof(int));
    hipMemset(d_buf, 0, buf_len * sizeof(int));

    int* d_write_count;
    hipMalloc(&d_write_count, sizeof(int));
    hipMemset(d_write_count, 0, sizeof(int));

    hiprandState* devStates;
    hipMalloc (&devStates, nodes_per_rank * sizeof(hiprandState));

    test_write_kernel<<<1, buf_len>>>(d_buf);

    int threads = 1024;
    int blocks = (nodes_per_rank + threads - 1)/threads;
    int seed = 39483 + 2 << rank;
    float p = h_p;

    // Setup the random number generator thing idk. 
    setup_kernel<<<blocks, threads>>>(devStates, seed);

    // Generate the graph.
    generate_graph_kernel<<<blocks, threads>>>(d_buf, d_write_count, buf_len, devStates, p, 
        node_count, nodes_per_rank, start_node, end_node);

    // Copy device memory to host memory.
    hipMemcpy(h_buf, d_buf, buf_len * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_write_count, d_write_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();



    hipFree(d_buf);
    hipFree(devStates);
}
