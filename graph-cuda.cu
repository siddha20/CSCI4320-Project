
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "hiprand/hiprand_kernel.h"

#define WORD_LENGTH 5

int node_count;
int size;
int rank;

__device__ int mutex = 0;

void cuda_init(int l_rank, int l_size, int l_node_count) {

    node_count = l_node_count;
    size = l_size;
    rank = l_rank;

    int device_count;
    hipError_t cE;
    
    // Get cuda device count
    if((cE = hipGetDeviceCount(&device_count)) != hipSuccess) {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, device_count);
        exit(-1);
    }

    // Assign rank to cuda device.
    if((cE = hipSetDevice(rank % device_count)) != hipSuccess) {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", rank, (rank % device_count), cE);
        exit(-1);
    }
    
    printf("Rank %d attached to CUDA device %d.\n", rank, (rank % device_count));

}

__global__ void test_write_kernel(int* data) {
    data[threadIdx.x] = threadIdx.x;
}

__global__ void setup_kernel(hiprandState* state, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, state + idx);
}

__global__ void generate_graph_kernel(int* data, int* write_count, int buf_len, hiprandState* state, float p, 
                                      int node_count, int nodes_per_rank, int start_node, int end_node) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = start_node + idx;

    // printf("write count: %d, buf_len %d\n", *write_count, buf_len);
    if (*write_count >= buf_len) return;

    if (i < end_node) {
        // Create edges i --> j
        for (int j = 0; j < node_count; j++) {
            float rand = hiprand_uniform(state + idx);
            if (rand <= p) {
                int old = 1;
                while (old) {
                    old = atomicCAS(&mutex, 0, 1); // lock
                    if (old == 0){
                        data[(*write_count)] = i;
                        data[(*write_count) + 1] = j;
                        // printf("edge %d --> %d, %f\n", data[(*write_count)], data[(*write_count) + 1], rand);
                        (*write_count) += 2;
                        // printf("write count %d\n", *write_count);
                        __threadfence();

                        atomicExch(&mutex, 0); // unlock
                    }
                }
            }
        }
        __syncthreads();
    }
}

__global__ void generate_graph_kernel_v2(int* data, int* write_count, int buf_len, hiprandState* state, float p, 
                                      int node_count, int nodes_per_rank, int start_node, int end_node) {
    
    // extern __shared__ int s[];
    // __shared__ int thread_write_count;
    // int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // int i = start_node + blockIdx.x;


    // int n = node_count/blockDim.x + ((node_count % blockDim.x) != 0);
    // int j_start = threadIdx.x * n;
    // int j_end = (j_start + n) < node_count ? j_start + n : node_count;

    // // if (threadIdx.x == 0) thread_write_count = 0;
    // // __syncthreads();

    // printf("j_start %d j_end %d\n", j_start, j_end);
    // for (int j = j_start; j < j_end; j++) {
    //     float rand = curand_uniform(state + idx);
    //     if (rand <= p) {
    //         // s[j] = 1;
    //     }
    //     printf("edge %d --> %d %d\n", i, j, 0);
    // }



    // __syncthreads();



}

void generate_graph(int* h_buf, int buf_len, int* h_write_count, float h_p) {

    int nodes_per_rank = (node_count/size) + ((node_count % size) != 0);
    int start_node = rank * nodes_per_rank;
    int end_node = (start_node + nodes_per_rank) < node_count ? start_node + nodes_per_rank : node_count;

    // Allocate memory on device. 
    int* d_buf;
    hipMalloc(&d_buf, buf_len * sizeof(int));
    hipMemset(d_buf, 0, buf_len * sizeof(int));

    int* d_write_count;
    hipMalloc(&d_write_count, sizeof(int));
    hipMemset(d_write_count, 0, sizeof(int));

    hiprandState* devStates;
    hipMalloc (&devStates, nodes_per_rank * sizeof(hiprandState));

    test_write_kernel<<<1, buf_len>>>(d_buf);

    int threads = 256;
    int blocks =  nodes_per_rank; // (nodes_per_rank + threads - 1)/threads;
    int seed = 39483 + 2 << rank;
    float p = h_p;
    int ratio = 1;


    // Setup the random number generator thing idk. 
    setup_kernel<<<blocks * ratio, threads>>>(devStates, seed);

    // Generate the graph.
    // generate_graph_kernel<<<blocks, threads>>>(d_buf, d_write_count, buf_len, devStates, p, 
    //     node_count, nodes_per_rank, start_node, end_node);

    // printf("here\n");
    // generate_graph_kernel_v2<<<blocks * ratio, threads>>>(d_buf, d_write_count, buf_len, devStates, p, 
    //     node_count, nodes_per_rank, start_node, end_node);
    // printf("here\n");

    // Copy device memory to host memory.
    hipMemcpy(h_buf, d_buf, buf_len * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_write_count, d_write_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();



    hipFree(d_buf);
    hipFree(devStates);
}
